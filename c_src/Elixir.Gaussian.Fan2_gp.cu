#include "hip/hip_runtime.h"
#include "erl_nif.h"

__global__
void fan2(float *m, float *a, float *b, int size, int t)
{
	int xidx = ((blockIdx.x * blockDim.x) + threadIdx.x);
	int yidx = ((blockIdx.y * blockDim.y) + threadIdx.y);
if(((xidx >= ((size - 1) - t)) || (yidx >= (size - t))))
{
return;
}

	a[(((size * ((xidx + t) + 1)) + yidx) + t)] = (a[(((size * ((xidx + t) + 1)) + yidx) + t)] - (m[((size * ((xidx + t) + 1)) + t)] * a[(((size * t) + yidx) + t)]));
if((yidx == 0))
{
	b[((xidx + t) + 1)] = (b[((xidx + t) + 1)] - (m[((size * ((xidx + t) + 1)) + t)] * b[t]));
}

}

extern "C" void fan2_call(ErlNifEnv *env, const ERL_NIF_TERM argv[], ErlNifResourceType* type)
  {

    ERL_NIF_TERM list;
    ERL_NIF_TERM head;
    ERL_NIF_TERM tail;
    float **array_res;

    const ERL_NIF_TERM *tuple_blocks;
    const ERL_NIF_TERM *tuple_threads;
    int arity;

    if (!enif_get_tuple(env, argv[1], &arity, &tuple_blocks)) {
      printf ("spawn: blocks argument is not a tuple");
    }

    if (!enif_get_tuple(env, argv[2], &arity, &tuple_threads)) {
      printf ("spawn:threads argument is not a tuple");
    }
    int b1,b2,b3,t1,t2,t3;

    enif_get_int(env,tuple_blocks[0],&b1);
    enif_get_int(env,tuple_blocks[1],&b2);
    enif_get_int(env,tuple_blocks[2],&b3);
    enif_get_int(env,tuple_threads[0],&t1);
    enif_get_int(env,tuple_threads[1],&t2);
    enif_get_int(env,tuple_threads[2],&t3);

    dim3 blocks(b1,b2,b3);
    dim3 threads(t1,t2,t3);

    list= argv[3];

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg1 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg2 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  enif_get_resource(env, head, type, (void **) &array_res);
  float *arg3 = *array_res;
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  int arg4;
  enif_get_int(env, head, &arg4);
  list = tail;

  enif_get_list_cell(env,list,&head,&tail);
  int arg5;
  enif_get_int(env, head, &arg5);
  list = tail;

   fan2<<<blocks, threads>>>(arg1,arg2,arg3,arg4,arg5);
    hipError_t error_gpu = hipGetLastError();
    if(error_gpu != hipSuccess)
     { char message[200];
       strcpy(message,"Error kernel call: ");
       strcat(message, hipGetErrorString(error_gpu));
       enif_raise_exception(env,enif_make_string(env, message, ERL_NIF_LATIN1));
     }
}
